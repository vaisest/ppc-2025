#include <cmath>
#include <memory>
#include <vector>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <ranges>
#include <numeric>
/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

__global__ void sum_kernel(const float *data, float *row_sums, const int ny, const int nx, const bool square)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if (row >= ny)
    {
        return;
    }
    for (int i = row; i < ny; i += stride)
    {
        float sum = 0.0;
        for (int x = 0; x < nx; x++)
        {
            const float val = data[i * nx + x];
            sum += square ? val * val : val;
        }
        row_sums[i] = square ? sqrt(sum) : sum / nx;
    }
}
__global__ void mean_sub_kernel(float *data, float *row_sums, const int ny, const int nx)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < ny * nx; i += stride)
    {
        int row = i / nx;
        data[i] -= row_sums[row];
    }
}
__global__ void sqrt_div_kernel(float *data, const float *row_sums, const int ny, const int nx)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < ny * nx; i += stride)
    {
        int row = i / nx;
        data[i] /= row_sums[row];
    }
}
__global__ void transpose_copy(float *data, float *transpose, const int ny, const int nx)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < ny * nx; i += stride)
    {
        int y = i / nx; // row
        int x = i % nx; // column
        transpose[x * ny + y] = data[i];
    }
}
__global__ void matmul_transpose_kernel(float *data, float *result, const int ny, const int nx)
{
    constexpr size_t M = 8;
    constexpr size_t N = 4;
    int y = (blockIdx.x * blockDim.x + threadIdx.x) * M;
    int x = (blockIdx.y * blockDim.y + threadIdx.y) * M;
    if (y >= ny || x >= ny)
    {
        return;
    }

    float stuff[M][M] = {};
    // const auto rem = nx % 16;
    // limit iteration length to within bounds
    const int iM = M > ny - y ? ny - y : M;
    const int jM = M > ny - x ? ny - x : M;
    if (y <= x) [[likely]]
    {
        const auto rem = nx % N;
        for (size_t k = 0; k < nx - rem; k += N)
        {
            for (size_t i = 0; i < iM; i++)
            {
                float ys[N];
                for (size_t v = 0; v < N; v++)
                {
                    ys[v] = data[(k + v) * ny + (y + i)];
                }
                for (size_t j = 0; j < jM; j++)
                {
                    for (size_t v = 0; v < N; v++)
                    {
                        stuff[i][j] += ys[v] * data[(k + v) * ny + (x + j)];
                    }
                }
            }
        }
        for (size_t k = nx - rem; k < nx; k++)
        {
            for (size_t i = 0; i < iM; i++)
            {
                for (size_t j = 0; j < jM; j++)
                {
                    stuff[i][j] += data[k * ny + (y + i)] * data[k * ny + (x + j)];
                }
            }
        }
    }
    for (size_t i = 0; i < iM; i++)
    {
        for (size_t j = 0; j < jM; j++)
        {
            result[(y + i) * ny + x + j] = stuff[i][j];
        }
    }
}
int div_up(int a, int b)
{
    return (a + b - 1) / b;
}

void correlate(int ny, int nx, const float *data, float *result)
{
    // input data
    float *dataGPU = NULL;
    CHECK(hipMalloc(&dataGPU, ny * nx * sizeof(float)));
    float *transposedGPU = NULL;
    CHECK(hipMalloc(&transposedGPU, ny * nx * sizeof(float)));
    // row sums, and squared sums used for normalising input
    float *rowsGPU = NULL;
    CHECK(hipMalloc(&rowsGPU, ny * sizeof(float)));
    // output
    float *resGPU = NULL;
    CHECK(hipMalloc(&resGPU, ny * ny * sizeof(float)));

    // copy input
    CHECK(hipMemcpy(dataGPU, data, ny * nx * sizeof(float), hipMemcpyHostToDevice));

    int block_size = 1024;
    int num_blocks = div_up(ny * nx, block_size);
    int num_blocks_ny = div_up(ny, block_size);
    // get average of rows
    sum_kernel<<<num_blocks_ny, block_size>>>(dataGPU, rowsGPU, ny, nx, false);
    CHECK(hipGetLastError());

    // subtract average of rows from each row
    mean_sub_kernel<<<num_blocks, block_size>>>(dataGPU, rowsGPU, ny, nx);
    CHECK(hipGetLastError());

    // get sum of squares of rows
    sum_kernel<<<num_blocks_ny, block_size>>>(dataGPU, rowsGPU, ny, nx, true);
    CHECK(hipGetLastError());

    // divide rows by sum of squares of rows
    sqrt_div_kernel<<<num_blocks, block_size>>>(dataGPU, rowsGPU, ny, nx);
    CHECK(hipGetLastError());

    // transpose. seems to only take 0.016380 s on the test server for
    // 12000x12000 and speeds up matrix multiplication quite a bit
    transpose_copy<<<num_blocks, block_size>>>(dataGPU, transposedGPU, ny, nx);
    CHECK(hipGetLastError());

    // performance at larger block sizes seems worse for some reason

    int size = 16;
    dim3 block(size, size);
    dim3 grid(div_up(ny, size), div_up(ny, size));

    // Test server M and N:
    // (4, 2): 1.62701 sec
    // (4, 4): 1.62746 sec
    // (4, 8): 1.6904 sec
    // (8, 2): 1.44815 sec
    // (8, 4): 1.47318 sec
    // (8, 6): 5.05893 sec
    // (8, 8): 4.16878 sec
    matmul_transpose_kernel<<<grid, block>>>(transposedGPU, resGPU, ny, nx);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(result, resGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));

    CHECK(hipFree(dataGPU));
    CHECK(hipFree(resGPU));
    CHECK(hipFree(rowsGPU));
    CHECK(hipFree(transposedGPU));
}
