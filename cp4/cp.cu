#include <cmath>
#include <memory>
#include <vector>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

__global__ void sum_kernel(const float *data, float *row_sums, const int ny, const int nx, const bool square)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if (row >= ny)
    {
        return;
    }
    for (int i = row; i < ny; i += stride)
    {
        float sum = 0.0;
        for (int x = 0; x < nx; x++)
        {
            const float val = data[i * nx + x];
            sum += square ? val * val : val;
        }
        row_sums[i] = square ? sqrt(sum) : sum / nx;
    }
}
__global__ void mean_sub_kernel(float *data, float *row_sums, const int ny, const int nx)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < ny * nx; i += stride)
    {
        int row = i / nx;
        data[i] -= row_sums[row];
    }
}
__global__ void sqrt_div_kernel(float *data, const float *row_sums, const int ny, const int nx)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < ny * nx; i += stride)
    {
        int row = i / nx;
        data[i] /= row_sums[row];
    }
}
__global__ void matmul_kernel(float *data, float *result, const int ny, const int nx)
{
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    if (y >= ny || x >= ny)
    {
        return;
    }

    float sum = 0.0;
    if (y <= x) [[likely]]
    {
        for (int i = 0; i < nx; i++)
        {
            sum += data[y * nx + i] * data[x * nx + i];
        }
    }
    result[y * ny + x] = sum;
}
int div_up(int a, int b)
{
    return (a + b - 1) / b;
}
void correlate(int ny, int nx, const float *data, float *result)
{
    // input data
    float *dataGPU = NULL;
    CHECK(hipMalloc(&dataGPU, ny * nx * sizeof(float)));
    // row sums, and squared sums used for normalising input
    float *rowsGPU = NULL;
    CHECK(hipMalloc(&rowsGPU, ny * sizeof(float)));
    // output
    float *resGPU = NULL;
    CHECK(hipMalloc(&resGPU, ny * ny * sizeof(float)));

    // copy input
    CHECK(hipMemcpy(dataGPU, data, ny * nx * sizeof(float), hipMemcpyHostToDevice));

    int block_size = 1024;
    int num_blocks = div_up(ny * nx, block_size);
    int num_blocks_ny = div_up(ny, block_size);
    // get average of rows
    sum_kernel<<<num_blocks_ny, block_size>>>(dataGPU, rowsGPU, ny, nx, false);
    CHECK(hipGetLastError());

    // subtract average of rows from each row
    mean_sub_kernel<<<num_blocks, block_size>>>(dataGPU, rowsGPU, ny, nx);
    CHECK(hipGetLastError());

    // get sum of squares of rows
    sum_kernel<<<num_blocks_ny, block_size>>>(dataGPU, rowsGPU, ny, nx, true);
    CHECK(hipGetLastError());

    // divide rows by sum of squares of rows
    sqrt_div_kernel<<<num_blocks, block_size>>>(dataGPU, rowsGPU, ny, nx);
    CHECK(hipGetLastError());

    // performance at larger block sizes seems worse for some reason
    int block_size_2d = 16;
    dim3 block2dSize(block_size_2d, block_size_2d);
    dim3 grid(div_up(ny, block_size_2d), div_up(ny, block_size_2d));

    // calculate AA^T
    matmul_kernel<<<grid, block2dSize>>>(dataGPU, resGPU, ny, nx);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(result, resGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));

    CHECK(hipFree(dataGPU));
    CHECK(hipFree(resGPU));
    CHECK(hipFree(rowsGPU));
}
